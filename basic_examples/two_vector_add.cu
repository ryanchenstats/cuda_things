// this is a C++ program that uses CUDA to run on the GPU
// nvcc two_vector_add.cu -o add_two_vectors
// produces a.out
// run with ./add_two_vectors
// cuda does this faster than CPU
// takes less 1/3 the time of CPU

#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 10000000
#define ops_per_thread 10

using namespace std;

// this time, make each thread compute several elements of the array
__device__ void prints()
{
    printf("Block size: %d\n", blockDim.x);
    printf("Block index: %d\n", blockIdx.x);
    printf("Thread index: %d\n", threadIdx.x);
    printf("Thread size: %d\n", blockDim.x);
    printf("Grid size: %d\n", gridDim.x);
}

__global__ void add(int *a, int *b, int *c)
{
    // index takes into account number of blocks and threads
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // gridDim.x is the number of blocks in the grid
    // blockDim.x is the number of threads in a block
    // stride is the total number of threads in a grid

    // our array is arranged in a linear fashion of
    // size N <= num_grids * gridDim.x * blockDim.x
    // iterate N by index
    // we skip ever blockdim * griddim elemts
    // basically says, we add every thread i out of ("stride") of
    //      grid[m] for each m
    // int flag = 0;
    for (int i = index; i < N; i += stride)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int *h_a, *h_b, *h_c;

    // perks using C++, no need to write to and from device
    // cudaMallocManaged uses Unifed Memory
    hipMallocManaged(&h_a, N * sizeof(int));
    hipMallocManaged(&h_b, N * sizeof(int));
    hipMallocManaged(&h_c, N * sizeof(int));

    for (int i = 0; i < N; i++)
    {
        h_a[i] = rand() % 10;
        h_b[i] = rand() % 10;
    }

    clock_t d_begin = clock();

    int num_blocks, size_of_block;
    size_of_block = 64;
    num_blocks = (N + size_of_block - 1) / size_of_block;

    // define num_grids, num_blocks, and threads per block
    add<<<1, num_blocks, size_of_block>>>(h_a, h_b, h_c);
    hipDeviceSynchronize();

    clock_t d_end = clock();

    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_c);

    hipMallocManaged(&h_a, N * sizeof(int));
    hipMallocManaged(&h_b, N * sizeof(int));
    hipMallocManaged(&h_c, N * sizeof(int));

    clock_t g_begin = clock();
    int num_grids = 2;
    num_blocks = (N + 1) / 2;
    // num grids, num blocks, shared memory
    add<<<num_blocks, size_of_block>>>(h_a, h_b, h_c);
    hipDeviceSynchronize();
    clock_t g_end = clock();

    // for (int i = 0; i < N; i++)
    // {
    //     printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    // }

    for (int i = 0; i < N; i++)
    {
        h_a[i] = rand() % 10;
        h_b[i] = rand() % 10;
    }

    clock_t h_begin = clock();
    for (int i = 0; i < N; i++)
    {
        h_c[i] = h_a[i] + h_b[i];
    }
    clock_t h_end = clock();

    cout << "Time taken by GPU 1 grid: " << (double)(d_end - d_begin) / CLOCKS_PER_SEC << endl;
    cout << "Time taken by GPU 2 grids: " << (double)(g_end - g_begin) / CLOCKS_PER_SEC << endl;
    cout << "Time taken by CPU: " << (double)(h_end - h_begin) / CLOCKS_PER_SEC << endl;

    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_c);

    return 0;
}
